/*
 * Russell Taylor(rtaylor)
 * Matt Crusse(macrusse)
 * CPE458-01 Lab 1 Winter 2013 
 */


#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include <sys/mman.h> 
#include <errno.h>
#include <string.h>
#include <stdarg.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <stdint.h>

#define TILE_SIZE 32

/*Compile-Time Declaration on double or float usage*/
#ifdef DOUBLE
#define TYPEUSE double

#else
#define TYPEUSE float

#endif

/* 
 * Handles CUDA errors, taking from provided sample code on clupo site
 */

static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



/*Reads Input File and Returns Buffer of Contents*/
char* read_file(const char * file_name) 
{
  size_t size;
  char *buffer;
  FILE *fp;
  
  fp = fopen(file_name,"r");
  if(!fp) {
    fprintf (stderr, "Error opening input file.\n");
    exit (EXIT_FAILURE);    
  }

  fseek (fp, 0, SEEK_END);
  size = ftell(fp);
  rewind (fp);
  
  buffer = (char*) malloc (sizeof(char)*size);
  fread (buffer, 1, size, fp);
  fclose(fp);
  return buffer;
}

/*Calculate the Resultant Matrix from Multiplication*/
void calc_matrix(TYPEUSE *A, TYPEUSE *B, TYPEUSE *C, int Arow, int Acol, int Brow, int Bcol)
{
  uint64_t i, j, k;
  TYPEUSE sum = 0;
  for(i = 0; i < Arow; i++)//Iterate through Matrix B columnwise
  {
    for(j = 0; j < Bcol; j++)//Iterate through Matrix A rowwise
    {
        for(k = 0; k < Acol; k++)//Acol = Brow on valid Matrices
        {
          if(i >475)
            printf("");
          sum+= A[ i* (Acol) + k] * B[k * (Bcol) + j];
          
        }
        C[i *Acol + j] = sum;
        sum = 0;
    }
  }
}

/* Print matrix values to a file outputfile */
void output_matrix(const char * outputfile, TYPEUSE *matrix, int row, int col) 
{
  int i, j;

  FILE *ofp = fopen(outputfile, "w");
  if(!ofp){
    fprintf (stderr, "Error opening output file.\n");
    exit (EXIT_FAILURE);    
  }

  for(i = 0; i < row; i++) {
    for(j = 0; j < col; j++) {
      fprintf(ofp, "%.2f ",matrix[i*uint64_t(col) + j]);
    }  
    if(i < row-1){
      fprintf(ofp, "\n");
    }
  }
  fclose(ofp);
}


/*
 * Simply prints out the matrix to screen 
 */
void print_matrix(TYPEUSE *matrix, int row, int col) 
{
  int i, j;
  for(i = 0; i < row; i++) {
    for(j = 0; j < col; j++) {
      //printf("(%d,%d)", i, j);
      printf("%.2f ",matrix[i*col +j]);
    }  
    if(i < row-1){
      printf("\n");
    }
  }
  printf("\n");

}

/*Created a Matrix based on Buffered Input Information*/
TYPEUSE * read_matrix(int * rowCnt, int * colCnt, char * mapped)
{
  TYPEUSE value;  
  const char *delim_space = " ";
  char *token = NULL;  
  char *unconverted;
  int i, j, len;
  TYPEUSE *matrix;
  uint64_t bigiter;
  *colCnt = 0;
  *rowCnt = 0;

  
  /* Determine Col Count */
  i = 0;
  while(mapped[i] != '\n'){
    if(mapped[i] == '.') {
     (*colCnt)++;
    }
    i++;
  }  

  /* Determine Row Count */
  bigiter = 0;//For large file sizes, an int is too small to iterate through
  len = strlen(mapped);
  while(bigiter < len && mapped[bigiter] != '\0'){
    if((mapped[bigiter] == '\n') && (mapped[bigiter+1] != '\0') ) {
     (*rowCnt)++;
    }
    bigiter+=1;
  }
  (*rowCnt)++;

  /* Malloc the Matrix */
  if (( matrix = (TYPEUSE *) malloc((*rowCnt) * (*colCnt) * sizeof(TYPEUSE))) == NULL ) {
    printf("malloc issue");
  }
    
  /* Read values into matrix */
  i = 0; j = 0;
  for (token = strtok(mapped, delim_space); token != NULL; token = strtok(NULL, delim_space)) {
    value = strtod(token, &unconverted);
    matrix[i*(*colCnt) +j] = value;
    j++;
    if(j == (*colCnt)) {
      j = 0;
      if(++i == (*rowCnt))
	      break;
    }
  }
  return matrix;

}

__device__ void copyMiniMatrix(TYPEUSE * M_device, TYPEUSE M_shared[TILE_SIZE][TILE_SIZE], uint64_t row, uint64_t col)
{
  if(threadIdx.y < TILE_SIZE && threadIdx.x < TILE_SIZE) {
    M_shared[threadIdx.y][threadIdx.x] = M_device[row + col];  
  }

}

__global__ void MMKernel(TYPEUSE *A_d, TYPEUSE *B_d, TYPEUSE * C_d, uint64_t depth, uint64_t Arow, uint64_t Bcol)
{
  TYPEUSE Cvalue = 0.0;
  __shared__ TYPEUSE A_shared[TILE_SIZE][TILE_SIZE], B_shared[TILE_SIZE][TILE_SIZE]; 
  int resultWidth = Bcol;
  int resultCol = blockIdx.x * blockDim.x + threadIdx.x;
  int resultRow = blockIdx.y * blockDim.y + threadIdx.y;  
  int resultIndex = resultRow * resultWidth + resultCol;

  /* Boundary check */
  if(resultRow >= Arow || resultCol >= Bcol) {
    return;
  }
  
  for(int i = 0; i < (depth+TILE_SIZE-1)/TILE_SIZE; i++)
  {
  
    /* Copy global matricies into shared memory */
    if(threadIdx.x + i* TILE_SIZE < depth){
      copyMiniMatrix(A_d, A_shared, resultRow*Arow, threadIdx.x + (i * TILE_SIZE));
    }
    if(threadIdx.y + i* TILE_SIZE < depth) {
      copyMiniMatrix(B_d, B_shared, (threadIdx.y + (i * TILE_SIZE))*Bcol, resultCol);
    }
    
    /* Wait for all threads to complete copy to shared memory */
    __syncthreads();


    for(int k = 0; k < TILE_SIZE; k++)
    {
      if(k + (i * TILE_SIZE) < depth) /* Boundary check */
      {
        TYPEUSE Aelem = A_shared[threadIdx.y][k];
        TYPEUSE Belem = B_shared[k][threadIdx.x];
        Cvalue += Aelem * Belem;
      }
    }

    /* Wait for all threads to finish */
    __syncthreads();
  }
    
  C_d[resultIndex] = Cvalue;
}

int main (int argc, const char * argv[])
{
  const char * Cfile = "result.out";
  TYPEUSE * Amatrix, * Bmatrix, * Cmatrix;
  TYPEUSE * A_d, * B_d, * C_d;
  int Arow, Acol, Brow, Bcol;
  int size;
  int blockRow, blockCol;
  char * Amapped, * Bmapped;

  if(argc != 3) { 
    fprintf(stderr, "Usage: [Matrix A] [Matrix B]\n");
    exit(EXIT_FAILURE);
  }

  /* Device Properties */
  /*
  cudaDeviceProp prop;
  cudaGetDeviceProperties(&prop,0);
  printf("maxThreads: %d\n", prop.maxThreadsPerBlock);
  */

  /* Read and Map matrix */
  Amapped = read_file(argv[1]);
  Bmapped = read_file(argv[2]);
  Amatrix = read_matrix(&Arow, &Acol, Amapped); 
  Bmatrix = read_matrix(&Brow, &Bcol, Bmapped);
  if(Acol != Brow) {
    fprintf(stderr, "Matrices are not a compatible size to be multiplied\n");
    exit(EXIT_FAILURE);
  }
  
  /* Malloc a New Matrix */
  if (( Cmatrix = (TYPEUSE *) malloc((Arow) * (Bcol) * sizeof(TYPEUSE))) == NULL ) {
    printf("malloc issue");
  }
  
  /* Malloc and Copy space on GPU */
  size = Arow * Acol * sizeof(TYPEUSE);
  HANDLE_ERROR(hipMalloc(&A_d, size));
  HANDLE_ERROR(hipMemcpy(A_d, Amatrix, size, hipMemcpyHostToDevice));
  
  size = Brow * Bcol * sizeof(TYPEUSE);
  HANDLE_ERROR(hipMalloc(&B_d, size));
  HANDLE_ERROR(hipMemcpy(B_d, Bmatrix, size, hipMemcpyHostToDevice));

  size = Arow * Bcol * sizeof(TYPEUSE);
  HANDLE_ERROR(hipMalloc(&C_d, size));
  
  blockRow = (Arow+31) / 32;
  blockCol = (Bcol+31) / 32;
  
  //printf("blockRow: %d\t blockCol: %d\n",blockRow,blockCol);
    
  /*Kernel Call*/
  dim3 dimGrid(blockCol,blockRow);
  dim3 dimBlock(32,32);
  MMKernel<<<dimGrid,dimBlock>>>(A_d, B_d, C_d, Brow, Arow, Bcol);

  HANDLE_ERROR(hipMemcpy(Cmatrix,C_d,size, hipMemcpyDeviceToHost));

  //output_matrix(Cfile, Cmatrix, Arow, Bcol);
  
  print_matrix(Cmatrix, Arow, Bcol);
  
  /* Free Stuff */
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  free(Amatrix);
  free(Bmatrix);
  free(Cmatrix);
  free(Amapped);
  free(Bmapped);

  return 0;
}
